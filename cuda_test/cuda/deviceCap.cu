#include "hip/hip_runtime.h"
#include "./common.h"

void listDevice(void)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    printf("device count %d \n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, i);


        printf("device name : %s\n", prop.name);
	printf("device major %d minor %d\n", prop.major, prop.minor);
	printf("warp size %d\n", prop.warpSize);
	printf("multi processor count %d\n", prop.multiProcessorCount);
	printf("max grid size %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("max thread dims %d %d %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("max thread count per block %d\n", prop.maxThreadsPerBlock);
	printf("max thread per multiprocessor %d\n", prop.maxThreadsPerMultiProcessor);
	printf("clock rate %d\n", prop.clockRate);

    }

}

void deviceCap(void)
{
    listDevice();

}