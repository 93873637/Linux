#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 128

const int size_x = 128;
const int size_y = 1;

__global__ static void threadDemo(unsigned int * ret)
{
    unsigned int xIndex = blockDim.x *blockIdx.x + threadIdx.x;
    unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

    printf("block Dim (%d, %d) \n", blockDim.x, blockDim.y);
    printf("block id (%d, %d)\n", blockIdx.x, blockIdx.y);
    printf("thread id (%d, %d) \n", threadIdx.x, threadIdx.y);
    
    if (xIndex < size_x && yIndex < size_y) {
        unsigned int index = xIndex + size_x * yIndex;

        ret[index] = xIndex;
	ret[index + size_x * size_y] = yIndex;
    }

}

void ThreadDemo(void)
{
    unsigned int * ret = 0;
    unsigned int  host_ret[size_x*size_y*2] = {0};
    int i = 0;

    hipMalloc((void**) &ret, sizeof(unsigned int)*(size_x*size_y*2));

    dim3 grid(size_x / BLOCK_DIM, 1);
    dim3 block(BLOCK_DIM, 1, 1);


    
    threadDemo<<<grid,block>>>(ret);

    hipMemcpy(&host_ret, ret, sizeof(unsigned int)*(size_x*size_y*2), hipMemcpyDeviceToHost);


    for (i = 0; i < size_x*size_y; i++) {
        printf("(%u,%u)", host_ret[i], host_ret[size_x*size_y+i]);

    }

    hipFree(ret);

}