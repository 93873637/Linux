#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define WINDOW_WIDTH 9
#define WINDOW_HEIGHT 7
#define IMAGE_WIDTH 1280
#define IMAGE_HEIGHT 720

#define BLOCK_SIZE 128
#define LINES_PER_BLOCK 16

__global__ static void censusTransform(int width, int height, int * ret)
{
    int half_kw = WINDOW_WIDTH /2;
    int half_kh = WINDOW_HEIGHT /2 ;

    int tid = threadIdx.x;
    int x0 = blockIdx.x * (BLOCK_SIZE - WINDOW_WIDTH + 1) - half_kw;
    int y0 = blockIdx.y * LINES_PER_BLOCK;

    printf("block Dim (%d %d) (%d %d) \n", blockDim.x, blockDim.y, blockIdx.x, blockIdx.y);
    //printf("block id (%d %d) \n", blockIdx.x, blockIdx.y);
    for (int i = 0; i < WINDOW_HEIGHT; ++i) {
        int x = x0 + tid;
	int y = y0 - half_kh + i;
        int value = 0;
	
        if (0 <= x && x < IMAGE_WIDTH && 0 <= y && y < IMAGE_HEIGHT) {
	    value = x+y*IMAGE_WIDTH;
	    ret[i] = value / IMAGE_WIDTH;
	    ret[WINDOW_HEIGHT + i] = value % IMAGE_WIDTH;
            //printf("row %d  col %d \n", value / IMAGE_WIDTH, value % IMAGE_WIDTH);
        }



    }
    __syncthreads();
}

void censusDemo(void)
{
    printf("censusDemo +++++\n");
    int * ret = 0;
    int host_ret[WINDOW_HEIGHT*2] = {0};
    
    hipMalloc((void**) &ret, sizeof(int)*WINDOW_HEIGHT*2);
    
    int width_per_block = BLOCK_SIZE - WINDOW_WIDTH + 1;
    int height_per_block = LINES_PER_BLOCK;
    dim3 grid((IMAGE_WIDTH + width_per_block - 1) / width_per_block, (IMAGE_HEIGHT + height_per_block - 1) / height_per_block);
    //dim3 grid(1,1);
    dim3 block(BLOCK_SIZE);

    censusTransform<<<grid,block>>>(IMAGE_WIDTH, IMAGE_HEIGHT, ret);

    hipMemcpy(host_ret, ret, sizeof(int)*WINDOW_HEIGHT*2, hipMemcpyDeviceToHost);
    for (int i = 0; i < WINDOW_HEIGHT; i++) {
        printf("(%d  %d)", host_ret[i], host_ret[WINDOW_HEIGHT+i]);

    }

    hipFree(ret);

}
